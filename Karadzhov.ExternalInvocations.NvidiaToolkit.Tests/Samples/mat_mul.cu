#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hipblas.h"

extern "C" __declspec(dllexport) void __cdecl matrixMul(int m, int n, int k, float* a, float* b, float* c);

__global__ void devMatrivMultiply(int m, int n, int k, float* d_a, float* d_b, float* d_c)
{
	hipblasHandle_t cnpHandle;
	hipblasCreate(&cnpHandle);

	float* params = (float*)malloc(2*sizeof(float));
	params[0] = 1.0f;
	params[1] = 0.0f;

	hipblasSgemm(cnpHandle,
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				m, n, k,
				&params[0],
				d_a, m,
				d_b, k,
				&params[1],
				d_c, m);

	hipblasDestroy(cnpHandle);

	free(params);
}

void __cdecl matrixMul(int m, int n, int k, float* a, float* b, float* c)
{
	float *d_a, *d_b, *d_c;

	hipMalloc((void **)&d_a, m * k * sizeof(float));
	hipMalloc((void **)&d_b, k * n * sizeof(float));
	hipMalloc((void **)&d_c, m * n * sizeof(float));

	hipblasSetVector(m * k, sizeof(float), a, 1, d_a, 1);
	hipblasSetVector(k * n, sizeof(float), b, 1, d_b, 1);
	hipblasSetVector(m * n, sizeof(float), c, 1, d_c, 1);

	devMatrivMultiply<<<1, 1>>>(m, n, k, d_a, d_b, d_c);

	hipblasGetVector(m * n, sizeof(float), d_c, 1, c, 1);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
}